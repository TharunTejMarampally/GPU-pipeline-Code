
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// Define the neural network architecture
#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10

// Sigmoid activation function
__device__ float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

// Kernel function for neural network inference
__global__ void neuralNetworkInference(float *input, float *weightsInputHidden, float *weightsHiddenOutput, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute the hidden layer activations
    float hidden[HIDDEN_SIZE] = {0};
    for (int i = 0; i < HIDDEN_SIZE; i++) {
        float sum = 0;
        for (int j = 0; j < INPUT_SIZE; j++) {
            sum += input[j] * weightsInputHidden[j * HIDDEN_SIZE + i];
        }
        hidden[i] = sigmoid(sum);
    }

    // Compute the output layer activations
    for (int i = 0; i < OUTPUT_SIZE; i++) {
        float sum = 0;
        for (int j = 0; j < HIDDEN_SIZE; j++) {
            sum += hidden[j] * weightsHiddenOutput[j * OUTPUT_SIZE + i];
        }
        output[i] = sigmoid(sum);
    }
}

int main() {
    // Input data (example)
    float input[INPUT_SIZE];
    for (int i = 0; i < INPUT_SIZE; i++) {
        input[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Model weights (example)
    float weightsInputHidden[INPUT_SIZE * HIDDEN_SIZE];
    float weightsHiddenOutput[HIDDEN_SIZE * OUTPUT_SIZE];
    for (int i = 0; i < INPUT_SIZE * HIDDEN_SIZE; i++) {
        weightsInputHidden[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < HIDDEN_SIZE * OUTPUT_SIZE; i++) {
        weightsHiddenOutput[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    float *d_input, *d_weightsInputHidden, *d_weightsHiddenOutput, *d_output;
    hipMalloc((void **)&d_input, INPUT_SIZE * sizeof(float));
    hipMalloc((void **)&d_weightsInputHidden, INPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc((void **)&d_weightsHiddenOutput, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float));
    hipMalloc((void **)&d_output, OUTPUT_SIZE * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_input, input, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weightsInputHidden, weightsInputHidden, INPUT_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weightsHiddenOutput, weightsHiddenOutput, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Launch neural network inference kernel
    int blockSize = 256;
    int numBlocks = (OUTPUT_SIZE + blockSize - 1) / blockSize;
    neuralNetworkInference<<<numBlocks, blockSize>>>(d_input, d_weightsInputHidden, d_weightsHiddenOutput, d_output);

    // Copy result from device to host
    float output[OUTPUT_SIZE];
    hipMemcpy(output, d_output, OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Output the inference result
    std::cout << "Inference result: ";
    for (int i = 0; i < OUTPUT_SIZE; i++) {
        std::cout << output[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_input);
    hipFree(d_weightsInputHidden);
    hipFree(d_weightsHiddenOutput);
    hipFree(d_output);

    return 0;
}


// **************************************************************
// #include <iostream>
// #include <queue>

// #define N 100 // Maximum number of nodes in the graph

// __global__ void bfs(int *adjacencyList, int *queue, bool *visited, int *level, int *queueSize, int *outputQueue, int *outputQueueSize) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;

//     if (tid == 0) {
//         *queueSize = 0;
//         *outputQueueSize = 0;
//     }

//     __syncthreads();

//     if (tid == 0) {
//         queue[*queueSize] = 0; // Start BFS from node 0
//         atomicAdd(queueSize, 1);
//         visited[0] = true;
//         level[0] = 0;
//     }

//     __syncthreads();

//     while (*queueSize > 0) {
//         int vertex;
//         if (tid < *queueSize) {
//             vertex = queue[tid];
//         }

//         __syncthreads();

//         if (tid < *queueSize) {
//             atomicSub(queueSize, 1);

//             // Add the vertex to the output queue
//             int pos = atomicAdd(outputQueueSize, 1);
//             outputQueue[pos] = vertex;

//             // Visit neighbors of the vertex
//             for (int i = adjacencyList[vertex]; i < adjacencyList[vertex + 1]; i++) {
//                 int neighbor = adjacencyList[i];
//                 if (!visited[neighbor]) {
//                     visited[neighbor] = true;
//                     level[neighbor] = level[vertex] + 1;
//                     int queuePos = atomicAdd(queueSize, 1);
//                     queue[queuePos] = neighbor;
//                 }
//             }
//         }

//         __syncthreads();
//     }
// }

// int main() {
//     int adjacencyList[N + 1] = {0, 1, 2, 3, 4, 5}; // Example adjacency list
//     int queue[N];
//     bool visited[N] = {false};
//     int level[N];
//     int queueSize, outputQueueSize;
//     int outputQueue[N];

//     int *d_adjacencyList, *d_queue, *d_level, *d_outputQueue, *d_queueSize, *d_outputQueueSize;
//     bool *d_visited;

//     // Allocate device memory
//     cudaMalloc((void **)&d_adjacencyList, (N + 1) * sizeof(int));
//     cudaMalloc((void **)&d_queue, N * sizeof(int));
//     cudaMalloc((void **)&d_visited, N * sizeof(bool));
//     cudaMalloc((void **)&d_level, N * sizeof(int));
//     cudaMalloc((void **)&d_queueSize, sizeof(int));
//     cudaMalloc((void **)&d_outputQueue, N * sizeof(int));
//     cudaMalloc((void **)&d_outputQueueSize, sizeof(int));

//     // Copy data from host to device
//     cudaMemcpy(d_adjacencyList, adjacencyList, (N + 1) * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(d_visited, visited, N * sizeof(bool), cudaMemcpyHostToDevice);

//     // Launch BFS kernel
//     bfs<<<1, N>>>(d_adjacencyList, d_queue, d_visited, d_level, d_queueSize, d_outputQueue, d_outputQueueSize);

//     // Copy result from device to host
//     cudaMemcpy(outputQueue, d_outputQueue, N * sizeof(int), cudaMemcpyDeviceToHost);
//     cudaMemcpy(&outputQueueSize, d_outputQueueSize, sizeof(int), cudaMemcpyDeviceToHost);

//     // Output the BFS traversal
//     std::cout << "BFS Traversal: ";
//     for (int i = 0; i < outputQueueSize; i++) {
//         std::cout << outputQueue[i] << " ";
//     }
//     std::cout << std::endl;

//     // Free device memory
//     cudaFree(d_adjacencyList);
//     cudaFree(d_queue);
//     cudaFree(d_visited);
//     cudaFree(d_level);
//     cudaFree(d_queueSize);
//     cudaFree(d_outputQueue);
//     cudaFree(d_outputQueueSize);

//     return 0;
// }


// *************************************************
// __global__ void addKernelPTXv4(float4  *c, const float4 *a, const float4 *b)
// {
//     int i = threadIdx.x;
//     c[i].x = a[i].x + b[i].x;
//     c[i].y = a[i].y + b[i].y;
//     c[i].z = a[i].z + b[i].z;
//     c[i].w = a[i].w + b[i].w;
// }











// ************************************
// #include <iostream>
// #include <stdio.h>

// // Matrix dimensions
// #define N 32
// #define BLOCK_SIZE 16

// // Kernel function to perform matrix multiplication
// __global__ void matrixMul(int *a, int *b, int *c) {
//     // Calculate the row and column indices of the element
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;

//     // Perform the matrix multiplication for the element in c[row][col]
//     int sum = 0;
//     for (int i = 0; i < N; i++) {
//         sum += a[row * N + i] * b[i * N + col];
//     }

//     // Store the result in c[row][col]
//     c[row * N + col] = sum;
// }

// int main() {
//     // Initialize matrices
//     int *a, *b, *c;
//     int size = N * N * sizeof(int);

//     // Allocate memory on the host
//     a = (int *)malloc(size);
//     b = (int *)malloc(size);
//     c = (int *)malloc(size);

//     // Initialize matrices with random values
//     for (int i = 0; i < N * N; i++) {
//         a[i] = rand() % 10;
//         b[i] = rand() % 10;
//     }

//     // Allocate memory on the device
//     int *d_a, *d_b, *d_c;
//     cudaMalloc((void **)&d_a, size);
//     cudaMalloc((void **)&d_b, size);
//     cudaMalloc((void **)&d_c, size);

//     //
